#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"

#define THREADS 512

__global__ void instance_iou_cuda_kernel(
    int64_t total_gt_instances, const int64_t* __restrict__ nInstance, int nProposal,
    const int64_t* __restrict__ proposals_idx, const int64_t* __restrict__ proposals_offset,
    const int64_t* __restrict__ instance_labels,
    const int64_t* __restrict__ offset_num_gt_instances, const int64_t* __restrict__ batch,
    const int64_t* __restrict__ instance_pointnum, float* proposals_iou)
{
    for (int proposal_id = blockIdx.x; proposal_id < nProposal; proposal_id += gridDim.x)
    {
        int start = proposals_offset[proposal_id];
        int end = proposals_offset[proposal_id + 1];
        int sampleIdx = batch[proposals_idx[start]];
        int sampleNInstances = nInstance[sampleIdx];
        int instanceOffset = offset_num_gt_instances[sampleIdx];
        int proposal_total = end - start;
        for (int instance_id = threadIdx.x; instance_id < sampleNInstances;
             instance_id += blockDim.x)
        {
            int instance_total = instance_pointnum[instanceOffset + instance_id];
            int intersection = 0;
            for (int i = start; i < end; i++)
            {
                int idx = proposals_idx[i];
                if ((int)instance_labels[idx] == instance_id + 1)
                { // 0 is reserved for "no instance"
                    intersection += 1;
                }
            }

            proposals_iou[instanceOffset + instance_id + proposal_id * total_gt_instances] =
                (float)intersection /
                ((float)(proposal_total + instance_total - intersection) + 1e-5);
        }
    }
}

// input: proposals_idx (sumNPoint), int
// input: proposals_offset (nProposal + 1), int
// input: instance_labels (N), int64_t, 0~total_nInst-1, -100
// input: instance_pointnum (total_nInst), int
// output: proposals_iou (nProposal, total_nInst), float
void instance_iou_kernel_wrapper(int64_t total_gt_instances, int64_t max_gt_instances,
                                 const int64_t* nInstance, int nProposal,
                                 const int64_t* proposals_idx, const int64_t* proposals_offset,
                                 const int64_t* instance_labels,
                                 const int64_t* offset_num_gt_instances, const int64_t* batch,
                                 const int64_t* instance_pointnum, float* proposals_iou)
{
    auto stream = at::cuda::getCurrentCUDAStream();
    instance_iou_cuda_kernel<<<std::min(nProposal, THREADS * THREADS),
                               std::min(max_gt_instances, (int64_t)THREADS), 0, stream>>>(
        total_gt_instances, nInstance, nProposal, proposals_idx, proposals_offset, instance_labels,
        offset_num_gt_instances, batch, instance_pointnum, proposals_iou);
}
